#include "hip/hip_runtime.h"
#include <stdio.h>     
#include <hipblas.h>        
#include "utils/utils.cuh"            
#define PPP 1
#include <hip/hip_runtime.h> 
#include <helper_functions.h> 
#include <hip/hip_runtime_api.h>
#include "kernels.cuh"      
#define multi 20   
int main(int argc, char **argv){                                 
// Iinitialization   
srand(10);      
int start_size = atoi(argv[1]);        
int end_size =  atoi(argv[2]);           
int test_size = end_size   ;
int MAX_SIZE = end_size;
int gap_size =  atoi(argv[3]);        
int st_kernel = atoi(argv[4]);  
int end_kernel = atoi(argv[5]);     
int kernel_number;
int num_tests = 20;                            
float alpha = 1.0;                      
float negative_1 = -1.0;                           
float beta = -1.5;                    
int max_size = max(end_size,test_size);        
int M, N, K;    
M = test_size; N = test_size;  K = test_size;
float *A = NULL, *B = NULL, *C_ref = NULL, *C = NULL, *E = NULL, *E_ = NULL, *Res=NULL, *error_injec = NULL;
float *check_A_col = NULL, *check_B_row = NULL, *check_C_col = NULL, *check_C_row = NULL, *check_A_row_mul_C=NULL, *check_B_row_mul_C=NULL;
float *dA = NULL,*dB = NULL, *dC_ref = NULL, *dC = NULL, *dE=NULL, *dE_ = NULL, *dRes =NULL, *derror_injec=NULL;
float *dcheck_A_col = NULL, *dcheck_B_row = NULL, *dcheck_C_col = NULL, *dcheck_C_row = NULL, *dcheck_A_col_mul_B=NULL, *dcheck_B_row_mul_A=NULL;
int size = max_size * sizeof (int);               
int deviceId;          
hipGetDevice(&deviceId);            
hipDeviceProp_t props = getDetails(deviceId);          

A = (float *)malloc(sizeof(float) * MAX_SIZE * MAX_SIZE);
error_injec = (float *)malloc(sizeof(float) * MAX_SIZE);
B = (float *)malloc(sizeof(float) * MAX_SIZE * MAX_SIZE);    
C = (float *)malloc(sizeof(float) * MAX_SIZE * MAX_SIZE);                                   
E = (float *)malloc(sizeof(float) * MAX_SIZE);  
E_ = (float *)malloc(sizeof(float) * MAX_SIZE);   
Res = (float *)malloc(sizeof(float) * 1);
check_A_col = (float *)malloc(sizeof(float) * MAX_SIZE); 
check_B_row = (float *)malloc(sizeof(float) * MAX_SIZE); 
check_C_col = (float *)malloc(sizeof(float) * MAX_SIZE);     
check_C_row = (float *)malloc(sizeof(float) * MAX_SIZE);
check_A_row_mul_C = (float *)malloc(sizeof(float) * MAX_SIZE);
check_B_row_mul_C = (float  *)malloc(sizeof(float) * MAX_SIZE);
            
C_ref = (float *)malloc(sizeof(float) * MAX_SIZE * MAX_SIZE);    
generate_random_matrix(A, MAX_SIZE);
generate_random_matrix(B, MAX_SIZE);                                                           
generate_random_matrix(C, MAX_SIZE);
fill_vector(Res, 0.0, 1);       
fill_vector(C, 0.0, MAX_SIZE * MAX_SIZE);          
fill_vector(error_injec, 1.0, MAX_SIZE);        
fill_vector(E, 1.0, MAX_SIZE);      
fill_vector(check_A_col, 0.0, MAX_SIZE);  
fill_vector(check_B_row, 0.0, MAX_SIZE); 
fill_vector(check_C_col, 0.0, MAX_SIZE);                                                    
fill_vector(check_C_row, 0.0, MAX_SIZE);  
fill_vector(check_A_row_mul_C, 0.0, MAX_SIZE);
fill_vector(check_B_row_mul_C, 0.0, MAX_SIZE);         
copy_matrix(C, C_ref, MAX_SIZE); 
for(int i = 1; i <= MAX_SIZE; ++i)E_[i] = (float)i; 
    
        
CUDA_CALLER(hipMalloc((void**) &dA, sizeof(float) * MAX_SIZE * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dB, sizeof(float) * MAX_SIZE * MAX_SIZE));  
CUDA_CALLER(hipMalloc((void**) &dC, sizeof(float) * MAX_SIZE * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(float) * MAX_SIZE * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &derror_injec, sizeof(float) * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dE, sizeof(float) * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dE_, sizeof(float) * MAX_SIZE)); 
CUDA_CALLER(hipMalloc((void**) &dcheck_A_col, sizeof(float) * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dcheck_B_row, sizeof(float) * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dcheck_C_col, sizeof(float) * MAX_SIZE)); 
CUDA_CALLER(hipMalloc((void**) &dcheck_C_row, sizeof(float) * MAX_SIZE));
CUDA_CALLER(hipMalloc((void**) &dcheck_A_col_mul_B, sizeof(float) * MAX_SIZE)); 
CUDA_CALLER(hipMalloc((void**) &dcheck_B_row_mul_A, sizeof(float) * MAX_SIZE));  
CUDA_CALLER(hipMalloc((void**) &dRes, sizeof(float)));   
CUDA_CALLER(hipMemcpy(dE, E, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));  
CUDA_CALLER(hipMemcpy(dE_, E_, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice)); 
CUDA_CALLER(hipMemcpy(dcheck_A_col, dcheck_A_col, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dcheck_B_row, check_B_row, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dcheck_C_col, check_C_col, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dcheck_C_row, check_C_row, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dcheck_A_col_mul_B, check_A_row_mul_C, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dcheck_B_row_mul_A, check_B_row_mul_C, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dRes, Res, sizeof(float), hipMemcpyHostToDevice));        
CUDA_CALLER(hipMemcpy(dA, A, sizeof(float) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice));     
CUDA_CALLER(hipMemcpy(dB, B, sizeof(float) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(dC, C, sizeof(float) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice));        
CUDA_CALLER(hipMemcpy(dC_ref, C, sizeof(float) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice));
CUDA_CALLER(hipMemcpy(derror_injec, error_injec, sizeof(float) * MAX_SIZE, hipMemcpyHostToDevice));
        
    
// Verification           
printf("Start verification!\n");
hipblasHandle_t handle;                  
hipblasCreate(&handle);                 
hipDeviceSynchronize(); 
beta = 0;                      
for(int i = st_kernel; i <= end_kernel; ++i){
    kernel_number = i;
for (int iter = 0; iter < 1 ; iter++){ 
hipblasSgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_T, M, N, K,  &alpha, dA, M, dB, N, &beta, dC_ref, M);
}
if(kernel_number == 1){                             
    dim3 blockDim(64);                        
    dim3 gridDim(CEIL_DIV(M, 16), CEIL_DIV(N, 16));      
    hipDeviceSynchronize(); 
    sgemm_small<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
}  
else if(kernel_number == 2){        
    dim3 blockDim(64);      
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));     
    hipDeviceSynchronize(); 
    sgemm_medium<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
}         
else if(kernel_number == 3){         
    dim3 blockDim(64);  
    dim3 gridDim(CEIL_DIV(M, 64), CEIL_DIV(N, 64)); 
    hipDeviceSynchronize();  
    sgemm_large<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
}  
else if(kernel_number == 4){ 
    dim3 blockDim(128);      
    dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 32));
    hipDeviceSynchronize();            
    sgemm_tall<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
}                 
else if(kernel_number == 5){                                   
    dim3 blockDim(128);  
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 128)); 
    hipDeviceSynchronize();   
    sgemm_wide<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
}                 
else if(kernel_number == 6){  
    for (int iter = 0; iter < 1; iter++){
    dim3 blockDim(256);    
    dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 128));
    hipDeviceSynchronize(); 
    sgemm_huge<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }      
}         
else if(kernel_number == 11){        
    for (int iter = 0; iter < 1; iter++){ 
    dim3 blockDim(64);  
    dim3 gridDim(CEIL_DIV(M, 16), CEIL_DIV(N, 16));
    hipDeviceSynchronize(); 
    ft_sgemm_small<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }
}  
else if(kernel_number == 12){
    for (int iter = 0; iter < 1; iter++){
    dim3 blockDim(64);  
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
    hipDeviceSynchronize();  
    ft_sgemm_medium<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }                
}      
else if(kernel_number == 13){   
    for (int iter = 0; iter < 1; iter++){  
    dim3 blockDim(64);  
    dim3 gridDim(CEIL_DIV(M, 64), CEIL_DIV(N, 64));
    hipDeviceSynchronize();       
    ft_sgemm_large<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }
} 
else if(kernel_number == 14){
    for (int iter = 0; iter < 1; iter++){     
    dim3 blockDim(128);                           
    dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 32));
    hipDeviceSynchronize(); 
    ft_sgemm_tall<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }   
}              
else if(kernel_number == 15){
    for (int iter = 0; iter < 1; iter++){
    dim3 blockDim(128);  
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N,  128));  
    hipDeviceSynchronize(); 
    ft_sgemm_wide<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }  
}  
else if(kernel_number == 16){
    for (int iter = 0; iter < 1; iter++){
    dim3 blockDim(256);  
    dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 128));
    hipDeviceSynchronize(); 
    ft_sgemm_huge<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);  
    }
}   
else{
    hipblasSgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_T, M, N, K, &alpha, dA, M, dB, N, &beta, dC, M);
} 
hipDeviceSynchronize();    
hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost);
hipDeviceSynchronize();
hipMemcpy(C_ref, dC_ref, sizeof(float) * M * N, hipMemcpyDeviceToHost);
hipDeviceSynchronize();                                                                    

if (!verify_matrix(C_ref, C, M, N)) { 
    printf("kernel %d failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n", kernel_number);
    // exit(-3);  
}    
fflush(stdout);              
printf("kernel %d finish verified!\n", kernel_number);      
hipDeviceSynchronize();  
}
// Performance Profiling    
printf("################## Performance (GFLOPS) ########################\n");
// printf("##################### kernel %d #########################\n", kernel_number);
// return 0; 
beta=-1.5;
int list[14] = {0, 1, 2, 3, 4, 5, 6, 10, 11, 12, 13, 14, 15, 16};
char arr[14][20] = {"cublas", "kernel_sgemm_small", "kernel_sgemm_medium", "kernel_sgemm_large", "kernel_sgemm_tall", "kernel_sgemm_wide", "kernel_sgemm_huge",
                    "abft_baseline", "abft_kernel_small", "abft_kernel_medium", "abft_kernel_large", "abft_kernel_tall", "abft_kernel_wide", "abft_kernel_huge"};
// return 0;  
printf("Matrix Size         |");
for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
printf("%8d|", max_size);
}    
printf("\n");
for(int jj = 0; jj < 14; ++jj){
    kernel_number = list[jj];       
    if(kernel_number < st_kernel)continue;
    if(kernel_number > end_kernel) break;                                                
    printf("%-20s|", arr[jj]);
    // CUDA_CALLER(hipMemcpy(dC, C, sizeof(float) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice));
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
    N = K = M = max_size;                                     
    hipEvent_t beg, end; 
    hipEventCreate(&beg);                        
    hipEventCreate(&end); 
    float elapsed = 0;       
    if (kernel_number == 0){     
        hipEventRecord(beg);                      
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();    
            hipblasSgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_N, M, N, K, &alpha, dA, M, dB, K, &beta, dC, M);
            hipDeviceSynchronize();  
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);  
    }    
    else if (kernel_number == 1){
        hipEventRecord(beg);
        dim3 blockDim(64);
        dim3 gridDim(CEIL_DIV(M, 16), CEIL_DIV(N, 16));
        
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            sgemm_small<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end);  
    }  
    else if (kernel_number == 2){ 
        hipEventRecord(beg);
        dim3 blockDim(64);
        dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            sgemm_medium<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        } 
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end);  
    }  
    else if (kernel_number == 3){     
        hipEventRecord(beg);                
        dim3 blockDim(64);                           
        dim3 gridDim(CEIL_DIV(M, 64), CEIL_DIV(N, 64));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize(); 
            sgemm_large<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize(); 
        }
        hipEventRecord(end);      
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    } 
    else if (kernel_number == 4){
        hipEventRecord(beg); 
        dim3 blockDim(128);
        dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 32)); 
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            sgemm_tall<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }   
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end);                  
    } 
    else if (kernel_number == 5){
        hipEventRecord(beg); 
        dim3 blockDim(128);
        dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 128));
        for(int ii = 0; ii < num_tests; ++ii){         
            hipDeviceSynchronize();
            sgemm_wide<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    } 
    else if (kernel_number == 6){
        hipEventRecord(beg);
        dim3 blockDim(256);
        dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 128));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            sgemm_huge<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    } 
    else if (kernel_number == 10){
        hipEventRecord(beg);
        baseline_ft_sgemm(num_tests, M,N, K, handle, dA, dB, dC, dE, dRes, dcheck_C_row, dcheck_C_col, dcheck_A_col_mul_B, dcheck_B_row_mul_A, dcheck_A_col, dcheck_B_row,  alpha, beta, negative_1);
        hipEventRecord(end);
        hipEventSynchronize(beg);  
        hipEventSynchronize(end); 
    }
    else if (kernel_number == 11){ 
        hipEventRecord(beg);
        dim3 blockDim(64);
        dim3 gridDim(CEIL_DIV(M, 16), CEIL_DIV(N, 16));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            ft_sgemm_small<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize(); 
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    }  
    else if (kernel_number == 12){ 
        hipEventRecord(beg);
        dim3 blockDim(64); 
        dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            ft_sgemm_medium<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end);  
    }  
    else if (kernel_number == 13){                                      
        hipEventRecord(beg);                                  
        dim3 blockDim(64);                       
        dim3 gridDim(CEIL_DIV(M, 64), CEIL_DIV(N, 64));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            ft_sgemm_large<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    } 
    else if (kernel_number == 14){
        hipEventRecord(beg);             
        dim3 blockDim(128);
        dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 32));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();
            ft_sgemm_tall<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);             
        hipEventSynchronize(end); 
    }  
    else if (kernel_number == 15){
        hipEventRecord(beg);
        dim3 blockDim(128);
        dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 128));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();        
            ft_sgemm_wide<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC, alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);      
        hipEventSynchronize(end); 
    } 
    else if (kernel_number == 16){
        hipEventRecord(beg);
        dim3 blockDim(256);   
        dim3 gridDim(CEIL_DIV(M, 128), CEIL_DIV(N, 128));
        for(int ii = 0; ii < num_tests; ++ii){
            hipDeviceSynchronize();   
                ft_sgemm_huge<<<gridDim, blockDim>>>(M, N, K, dA, dB, dC,  alpha, beta);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);     
        hipEventSynchronize(beg);
        hipEventSynchronize(end); 
    } 
    hipEventElapsedTime(&elapsed, beg, end);                     
    double gflops  = 0.;
    gflops = double(2 * num_tests * double(M) * double(N) * double(K)) / (1e9);
    double perf = gflops / (elapsed / 1e3);
    printf("%8.0f|", perf);
    fflush(stdout);
}
printf("\n");
}
}
